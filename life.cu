
#include <hip/hip_runtime.h>
#include <sys/types.h>
#include <sys/ioctl.h>
#include <sys/mman.h>
#include <linux/fb.h>
#include <fcntl.h>

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*
   Conway's Game of Life running on nVidia GPUs.

   Each GPU thread updates the state of a single cell in the playing field, then the current generation is
   displayed in Linux's framebuffer (this means the program must be run outside X Window System).
   Since this program is just a demonstration, the data model is NOT decoupled from its representation.
*/



/**
   Cell drawing function

   Updates the state of a single cell and draws it on screen.
   Each GPU thread updates a single cell (the mapping is linear, so even if the field's size exceeds the
   number of computation units of the GPU, the pixels assigned to the same thread are far away from each other),
   based on the number of neighbors.
   Since the number of neighbors is only read (and this function is called AFTER the neighbors array is written)
   and each thread writes only its cell, there are no race conditions.

   @param field Array of cell states. 1: live cell, 0: dead cell.
   @param neighbors Array of neighbors' count (e.g. neighbors[42] = 4 means cell 42 has 4 neighbors).
   @param screen Framebuffer (the function assumes the buffer uses 32 bit colors).
   @param w Width of the framebuffer.
   @param h Height of the framebuffer.
*/
__global__ void updateCell(char *field, char *neighbors, char *screen, int w, int h)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x; /* Pick the cell based on thread and block IDs. */
  
  if (i < w * h)
  {
    if ((field[i] != 0 && (neighbors[i] == 3 || neighbors[i] == 2)) || /* Survive rule: S23 */
	(field[i] == 0 && neighbors[i] == 3))                            /* Birth rule: B3    */
    {
      /* Live cell */
      field[i] = 1;
      screen[4 * i] =  0xff;     /* blue  */
      screen[4 * i + 1] =  0xff; /* green */
      screen[4 * i + 2] =  0xff; /* red   */
      screen[4 * i + 3] =  0xff; /* alpha */
    }
    else
    {
      /* Dead cell */
      field[i] = 0;
      screen[4 * i] =  0x00;
      screen[4 * i + 1] =  0x00;
      screen[4 * i + 2] =  0x00;
      screen[4 * i + 3] =  0x00;
    }
  }
}

/**
   Neighbors counting function

   Checks the eight neighbors' state and counts the live ones.
   Each GPU thread writes the neighbors count for a single cell while the field is not modified, so there
   are no race conditions.

   @param field Array of cell states. 1: live cell, 0: dead cell.
   @param neighbors Array of neighbors' count (e.g. neighbors[42] = 4 means cell 42 has 4 neighbors).
   @param w Width of the framebuffer.
   @param h Height of the framebuffer.
 */
__global__ void countNeighbors(char *field, char *neighbors, int w, int h)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int x = i % w;
  int y = i / w * w; /* Rounding to the closest multiple of w. */
  int xp1 = (i + 1) % w;
  int xm1 = (i + w - 1) % w;
  int yp1 = ((i + w) % (h * w)) / w * w;
  int ym1 = ((i - w + h * w) % (h * w)) / w * w;
   
  neighbors[i] = 0;
  if (field[xm1 + ym1] != 0)
    neighbors[i]++;
  if (field[xm1 + y] != 0)
    neighbors[i]++;
  if (field[xm1 + yp1] != 0)
    neighbors[i]++;
  if (field[x + ym1] != 0)
    neighbors[i]++;
  if (field[x + yp1] != 0)
    neighbors[i]++;
  if (field[xp1 + ym1] != 0)
    neighbors[i]++;
  if (field[xp1 + y] != 0)
    neighbors[i]++;
  if (field[xp1 + yp1] != 0)
    neighbors[i]++;
}

/**
   Game of Life wrapper function.

   Initializes the playing field with a random initial state and iterates over Life's generations.
   This function takes care of synchronizing GPU threads so that no race conditions are triggered.

   @param screen Framebuffer.
   @param w Width of the framebuffer.
   @param h Height of the framebuffer.
 */
void life(char *screen, int w, int h)
{
  char field[w * h];
  
  /* Generate a random initial playing field. */
  srand(time(NULL));
  for (int i = 0; i < w * h; i++)
    field[i] = (rand() % 100 < 75)? 1: 0;

  /* Find the maximum number of threads for the current GPU. */
  struct hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, 0);
  char *d_field, *d_screen, *d_neighbors;
  int maxthds = properties.maxThreadsPerBlock;

  /*
     Copy memory to the GPU's internal RAM. Since this process is slow, it's done only once and the state
     of the playing field is effectively kept in GPU's memory, updating the main RAM only to display the
     result on screen.
  */
  hipMalloc((void **) &d_field, sizeof(char) * w * h);
  hipMalloc((void **) &d_neighbors, sizeof(char) * w * h);
  hipMalloc((void **) &d_screen, sizeof(char) * w * h * 4);
  hipMemcpy(d_field, field, sizeof(char) * w * h, hipMemcpyHostToDevice);
  hipMemcpy(d_screen, screen, sizeof(char) * w * h * 4, hipMemcpyHostToDevice);
  
  
  while (1) /* Maybe a termination condition would have been more elegant... */
  {
    /*
       Count the neighbors, synchronize the threads (in order to avoid race conditions),
       then update the state of each cell.
    */
    countNeighbors<<<(w * h + maxthds - 1) / maxthds, maxthds>>>(d_field, d_neighbors, w, h);
    hipDeviceSynchronize();
    updateCell<<<(w * h + maxthds - 1) / maxthds, maxthds>>>(d_field, d_neighbors, d_screen, w, h);

    /*
       Synchronize the threads again and finally copy the framebuffer from GPU's internal memory to main RAM
       (and therefore display the result).
    */
    hipDeviceSynchronize();
    hipMemcpy(screen, d_screen, sizeof(char) * w * h * 4, hipMemcpyDeviceToHost);
  }
  
  
  /* Free the GPU's internal memory. Since the loop condition is never false, this cleanup is never performed. */
  hipFree(d_field);
  hipFree(d_neighbors);
  hipFree(d_screen);
  
  /* printf("%s\n", cudaGetErrorString(cudaGetLastError())); Debug message. */
}


/* Semplice funzione main che controlla il framebuffer. Copiata da: https://stackoverflow.com/a/1830865

/**
   Program entrypoint.

   Initializes the main framebuffer (/dev/fb0), as suggested on: https://stackoverflow.com/a/1830865
   then passes it to the life() function.

   @return 0 if no error occurred, 1 otherwise.
 */
int main()
{
  struct fb_var_screeninfo screen_info;
  struct fb_fix_screeninfo fixed_info;
  char *buffer = NULL;
  size_t buflen;
  int fd = -1;
  int r = 1;

  fd = open("/dev/fb0", O_RDWR);
  if (fd >= 0)
  {
    if (!ioctl(fd, FBIOGET_VSCREENINFO, &screen_info) &&
        !ioctl(fd, FBIOGET_FSCREENINFO, &fixed_info))
    {
      buflen = screen_info.yres_virtual * fixed_info.line_length;
      buffer = (char *) mmap(NULL, buflen,
                       PROT_READ|PROT_WRITE, MAP_SHARED, fd, 0);
      if (buffer != MAP_FAILED)
      {
	life(buffer, screen_info.xres_virtual, screen_info.yres_virtual); /* Start the game. */
	r = 0;
      }
      else
      {
	perror("mmap");
      }
    }
    else
      {
	perror("ioctl");
      }
   }
   else
   {
       perror("open");
   }

   if (buffer && buffer != MAP_FAILED)
     munmap(buffer, buflen);
   if (fd >= 0)
     close(fd);

   return r;
}
